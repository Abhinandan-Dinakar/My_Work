#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#define SIZE 1024

__global__ void VectorAdd(int* a, int* b, int* c, int* d, int n)
{
	int i = threadIdx.x;

	if (i < n)
		d[i] = a[i] + b[i] + c[i];
}


int main()
{
	int* a, * b, * c, * d;

	hipMallocManaged(&a, SIZE * sizeof(int));
	hipMallocManaged(&b, SIZE * sizeof(int));
	hipMallocManaged(&c, SIZE * sizeof(int));
	hipMallocManaged(&d, SIZE * sizeof(int));

	for (int i = 0; i < SIZE; ++i)
	{
		a[i] = i;
		b[i] = i;
		c[i] = i;
		d[i] = 0;
	}
	VectorAdd << <1, SIZE >> > (a, b, c, d, SIZE);

	hipDeviceSynchronize();


	for (int i = 0; i < 10; ++i)
		printf("d[%d] = %d\n", i, d[i]);

	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(d);

	return 0;
}