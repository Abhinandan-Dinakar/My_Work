﻿/*
 * Program Name: Three Vector Addition using CUDA
 * Description : This program performs element-wise addition of three vectors using CUDA parallel programming.
 * Author      : Abhinandan Dinakar
 * Date        : 25.02.2022
 * Compiler    : nvcc (NVIDIA CUDA Compiler)
 * 
 * Usage:
 * - This program allocates unified memory for three input vectors and one output vector.
 * - The CUDA kernel is launched with one block and multiple threads to perform addition in parallel.
 * - The first 10 elements of the resulting vector are printed.
 * 
 * Note:
 * - Ensure that the CUDA toolkit is installed and a compatible GPU is available.
 * - Compile using: nvcc filename.cu -o outputfile
 */



#include <hip/hip_runtime.h>
#include <stdio.h>
#define SIZE 1024  // Define the size of the vectors

// CUDA kernel function to perform element-wise addition of three vectors
__global__ void VectorAdd(int* a, int* b, int* c, int* d, int n)
{
    int i = threadIdx.x; // Get the thread index within the block

    // Ensure that the thread index is within the valid range
    if (i < n)
        d[i] = a[i] + b[i] + c[i];  // Perform element-wise addition of three vectors
}

int main()
{
    int* a, * b, * c, * d; // Pointers to store vectors in unified memory

    // Allocate unified memory (accessible by both CPU and GPU)
    hipMallocManaged(&a, SIZE * sizeof(int));
    hipMallocManaged(&b, SIZE * sizeof(int));
    hipMallocManaged(&c, SIZE * sizeof(int));
    hipMallocManaged(&d, SIZE * sizeof(int));

    // Initialize vectors a, b, and c, and set vector d to zero
    for (int i = 0; i < SIZE; ++i)
    {
        a[i] = i;  // Assign values from 0 to SIZE-1
        b[i] = i;  // Assign values from 0 to SIZE-1
        c[i] = i;  // Assign values from 0 to SIZE-1
        d[i] = 0;  // Initialize d to 0
    }

    // Launch the CUDA kernel with 1 block and SIZE threads
    VectorAdd<<<1, SIZE>>>(a, b, c, d, SIZE);

    // Wait for GPU computations to complete before accessing results
    hipDeviceSynchronize();

    // Print the first 10 elements of the result vector d
    for (int i = 0; i < 10; ++i)
        printf("d[%d] = %d\n", i, d[i]);

    // Free allocated memory
    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipFree(d);

    return 0;  // Exit the program
}
